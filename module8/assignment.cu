
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>


#define index(r,c,l) (((r)*(l))+(c))

static const int NO_OFFSET = 0;
static const int MAX = 100; // arbitrary value

__device__ unsigned int get_thread_index()
{
	return (blockIdx.x * blockDim.x) + threadIdx.x;
}

__global__ void init_random_number_states(unsigned int seed, hiprandState_t *states) {
    const unsigned int thread_idx = get_thread_index();

    hiprand_init(seed, thread_idx, NO_OFFSET, &states[thread_idx]);
}

__global__ void generate_random_numbers(hiprandState_t* states, unsigned int* numbers) {
    const unsigned int thread_idx = get_thread_index();
    numbers[thread_idx] = hiprand(&states[thread_idx]) % MAX;
}

int main(int argc, char **argv) {

    // TODO arguments


    int N = 10, M = 10;

    hiprandState_t* states;
    hipMalloc((void**) &states, N * M * sizeof(hiprandState_t));

    unsigned int* gpu_A;
    unsigned int* gpu_B;
    // FIXME can't maintain this!
    unsigned int* cpu_A = new unsigned int[N * M];
    unsigned int* cpu_B = new unsigned int[M * N];


    hipMalloc((void**) &gpu_A, N * M * sizeof(unsigned int));
    hipMalloc((void**) &gpu_B, M * N * sizeof(unsigned int));


    init_random_number_states<<<N, M>>>(0, states);
    generate_random_numbers<<<N, M>>>(states, gpu_A);
    generate_random_numbers<<<M, N>>>(states, gpu_B);

    hipMemcpy(cpu_A, gpu_A, N * M * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_B, gpu_B, N * M * sizeof(unsigned int), hipMemcpyDeviceToHost);



    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("[%u, %u] -> %u\n", i, j, cpu_A[i * M + j]);
        }
    }

    cublasInit();

    // this is what we want to run when we end up running cublas
    // cublasSgemm('n','n',HA,WB,WA,1,AA,HA,BB,HB,0,CC,HC);


    cublasShutdown();

    hipFree(states);
    hipFree(gpu_A);
    hipFree(gpu_B);

    delete[] cpu_A;
    delete[] cpu_B;


    hipfftDoubleReal *signal;
    hipfftDoubleComplex *freq_domain;
    hipfftHandle plan;

    int fft_size = 16;
    size_t real_data_size_bytes = fft_size * sizeof(hipfftDoubleReal);
    size_t complex_data_size_bytes = fft_size * sizeof(hipfftDoubleComplex);
    hipMalloc((void**) &signal, real_data_size_bytes);
    hipMalloc((void**)&freq_domain, complex_data_size_bytes);

    // TODO is batch supposed to be used if we need to perform an fft that's smaller than the data?
    hipfftPlan1d(&plan, fft_size, HIPFFT_D2Z, 1);

    // out-of-place because we are doing a real-optimized fft
    // could do an in-place if we were doing a complex->complex fft
    hipfftExecD2Z(plan, signal, freq_domain);

    // Perform FFT
    // cufftExecD2Z()

    hipfftDestroy(plan);
    hipFree(signal);
    hipFree(freq_domain);

    return EXIT_SUCCESS;
}