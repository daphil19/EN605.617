
#include <hip/hip_runtime.h>
#include <iostream>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>



#define index(r,c,l) (((r)*(l))+(c))

static const int NO_OFFSET = 0;
static const int MAX = 10; // arbitrary value

__device__ unsigned int get_thread_index()
{
	return (blockIdx.x * blockDim.x) + threadIdx.x;
}

__global__ void init_random_number_states(unsigned int seed, hiprandState_t *states) {
    const unsigned int thread_idx = get_thread_index();

    hiprand_init(seed, thread_idx, NO_OFFSET, &states[thread_idx]);
}

__global__ void generate_random_numbers(hiprandState_t* states, double* numbers) {
    const unsigned int thread_idx = get_thread_index();
    numbers[thread_idx] = hiprand(&states[thread_idx]) % MAX;
}

__global__ void generate_cosine_wave(hipfftDoubleReal* signal) {
    const unsigned int thread_idx = get_thread_index();
    // yes, this caues waprs, but we don't care as much about the performance
    // of this kernel as its just generating example data
    switch(thread_idx % 4) {
        case 0: signal[thread_idx] = 1; break;
        case 1: signal[thread_idx] = 0; break;
        case 2: signal[thread_idx] = -1; break;
        case 3: signal[thread_idx] = 0; break;
    }
    // signal[thread_idx] += 1;
}

int main(int argc, char **argv) {

    // TODO arguments


    int N = 10, M = 10;

//     // curandState_t* states;
//     // cudaMalloc((void**) &states, M*N * sizeof(curandState_t));


    double* cpu_A = new double[M*N];
    double* cpu_B = new double[N*M];
    double* cpu_C = new double[M*M];

    for (int i = 0; i < M * N; i++) {
        cpu_A[i] = rand() % MAX;
        cpu_B[i] = rand() % MAX;
    }

    for (int i = 0; i < M * M; i++) {
        cpu_C[i] = rand() % MAX;
    }




	// curandGenerator_t rng;
	// curandCreateGenerator(&rng, CURAND_RNG_PSEUDO_DEFAULT);

	// // // while this code is invoked from the host, it actually is run on device
	// curandGenerateUniformDouble(rng, cpu_A, M*N);
	// // curandGenerateUniformDouble(rng, cpu_B, N*M);

	// curandDestroyGenerator(rng);


    // for (int i = 0; i < M*N; i++) {
    //     std::cout << cpu_A[i] << std::endl;
    // }

    double* gpu_A;
    double* gpu_B;
    double* gpu_C;
    cublasInit();

    cublasAlloc(M*N, sizeof(double), (void**)&gpu_A);
    cublasAlloc(N*M, sizeof(double), (void**)&gpu_B);
    cublasAlloc(M*M, sizeof(double), (void**)&gpu_C);


    hipblasSetMatrix(M, N, sizeof(double), cpu_A, M, gpu_A, M);
    hipblasSetMatrix(N, M, sizeof(double), cpu_B, N, gpu_B, N);


// // FIXME the GPU guys need to 
    cublasDgemm('n', 'n', M, M, N, 1, gpu_A, M, gpu_B, N, 0, gpu_C, M);

    hipblasGetMatrix(M, M, sizeof(double), gpu_C, M, cpu_C, M);


//     // cudaFree(states);
    cublasFree(gpu_A);
    cublasFree(gpu_B);
    cublasFree(gpu_C);
    
    cublasShutdown();

    delete[] cpu_A;
    delete[] cpu_B;
    delete[] cpu_C;


    hipfftDoubleReal *signal;
    hipfftDoubleComplex *freq_domain;
    hipfftHandle plan;

    int fft_size = 4;
    size_t real_data_size_bytes = fft_size * sizeof(hipfftDoubleReal);


    // for real->complex fft, the result size is dataSize (read: fftSize) / 2 + 1
    int results_size = (fft_size / 2) + 1;

    size_t complex_data_size_bytes = results_size * sizeof(hipfftDoubleComplex);
    hipMalloc((void**) &signal, real_data_size_bytes);
    hipMalloc((void**)&freq_domain, complex_data_size_bytes);

    // TODO is this a good layout? or should we try to optimize better?
    generate_cosine_wave<<<fft_size, 1>>>(signal);

    // TODO is batch supposed to be used if we need to perform an fft that's smaller than the data?
    hipfftPlan1d(&plan, fft_size, HIPFFT_D2Z, 1);

    // out-of-place because we are doing a real-optimized fft
    // could do an in-place if we were doing a complex->complex fft
    hipfftExecD2Z(plan, signal, freq_domain);

    // int results_size = (fft_size / 2) + 1;

    double2 *result = new double2[results_size];

    hipMemcpy(result, freq_domain, complex_data_size_bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < results_size; i++) {
        std::cout << result[i].x << " " << result[i].y << std::endl;
    }

    hipfftDestroy(plan);
    hipFree(signal);
    hipFree(freq_domain);
    delete[] result;

    return EXIT_SUCCESS;
}
