
#include <hip/hip_runtime.h>
#include <iostream>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

#define index(r,c,l) (((r)*(l))+(c))

static const int NO_OFFSET = 0;
static const int MAX = 10; // arbitrary value

__device__ unsigned int get_thread_index()
{
	return (blockIdx.x * blockDim.x) + threadIdx.x;
}

__global__ void init_random_number_states(unsigned int seed, hiprandState_t *states) {
    const unsigned int thread_idx = get_thread_index();

    hiprand_init(seed, thread_idx, NO_OFFSET, &states[thread_idx]);
}

__global__ void generate_random_numbers(hiprandState_t* states, double* numbers) {
    const unsigned int thread_idx = get_thread_index();
    numbers[thread_idx] = hiprand(&states[thread_idx]) % MAX;
}

__global__ void generate_cosine_wave(hipfftDoubleReal* signal) {
    const unsigned int thread_idx = get_thread_index();
    // yes, this caues waprs, but we don't care as much about the performance
    // of this kernel as its just generating example data
    switch(thread_idx % 4) {
        case 0: signal[thread_idx] = 1; break;
        case 1: signal[thread_idx] = 0; break;
        case 2: signal[thread_idx] = -1; break;
        case 3: signal[thread_idx] = 0; break;
    }
}

int main(int argc, char **argv) {

    int N = 10, M = 10;

    std::cout << "Matrix Multiplication: (" << M << "x" << N << ") * (" << N << "x" << M << ")" << std::endl;

    double* cpu_A = new double[M*N];
    double* cpu_B = new double[N*M];
    double* cpu_C = new double[M*M];

    for (int i = 0; i < M * N; i++) {
        cpu_A[i] = rand() % MAX;
        cpu_B[i] = rand() % MAX;
    }

    for (int i = 0; i < M * M; i++) {
        cpu_C[i] = rand() % MAX;
    }

    double* gpu_A;
    double* gpu_B;
    double* gpu_C;
    cublasInit();

    cublasAlloc(M*N, sizeof(double), (void**)&gpu_A);
    cublasAlloc(N*M, sizeof(double), (void**)&gpu_B);
    cublasAlloc(M*M, sizeof(double), (void**)&gpu_C);

    hipblasSetMatrix(M, N, sizeof(double), cpu_A, M, gpu_A, M);
    hipblasSetMatrix(N, M, sizeof(double), cpu_B, N, gpu_B, N);


    cublasDgemm('n', 'n', M, M, N, 1, gpu_A, M, gpu_B, N, 0, gpu_C, M);

    hipblasGetMatrix(M, M, sizeof(double), gpu_C, M, cpu_C, M);

    cublasFree(gpu_A);
    cublasFree(gpu_B);
    cublasFree(gpu_C);
    
    cublasShutdown();


    // print output
    std::cout << "Matrix A:" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << cpu_A[index(i, j, M)] << " ";
        }
        std::cout << std::endl;
    }

    // extra \n for better separation
    std::cout << std::endl;

    std::cout << "Matrix B:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << cpu_B[index(i, j, N)] << " ";
        }
        std::cout << std::endl;
    }

    // extra \n for better separation
    std::cout << std::endl;

    std::cout << "Matrix C (A x B);" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << cpu_C[index(i, j, M)] << " ";
        }
        std::cout << std::endl;
    }

    delete[] cpu_A;
    delete[] cpu_B;
    delete[] cpu_C;


    hipfftDoubleReal *signal;
    hipfftDoubleComplex *freq_domain;
    hipfftHandle plan;

    int fft_size = 4;

    // extra \ns for better separation
    std::cout << std::endl << std::endl;

    std::cout << "Perform FFT of size " << fft_size << std::endl;

    size_t real_data_size_bytes = fft_size * sizeof(hipfftDoubleReal);

    // for real->complex fft, the result size is dataSize (read: fftSize) / 2 + 1
    int results_size = (fft_size / 2) + 1;

    size_t complex_data_size_bytes = results_size * sizeof(hipfftDoubleComplex);
    hipMalloc((void**) &signal, real_data_size_bytes);
    hipMalloc((void**)&freq_domain, complex_data_size_bytes);

    // TODO is this a good layout? or should we try to optimize better?
    generate_cosine_wave<<<fft_size, 1>>>(signal);

    // TODO is batch supposed to be used if we need to perform an fft that's smaller than the data?
    hipfftPlan1d(&plan, fft_size, HIPFFT_D2Z, 1);

    // out-of-place because we are doing a real-optimized fft
    // could do an in-place if we were doing a complex->complex fft
    hipfftExecD2Z(plan, signal, freq_domain);

    // int results_size = (fft_size / 2) + 1;

    double2 *result = new double2[results_size];

    std::cout << "Input Singal" << std::endl;
    double *signal_cpu = new double[fft_size];
    hipMemcpy(signal_cpu, signal, real_data_size_bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < fft_size; i++) {
        std::cout << signal_cpu[i] << std::endl;
    }  
    delete[] signal_cpu;

    // extra \n for better separation
    std::cout << std::endl;

    std::cout << "FFT output (dropping redundant data)" << std::endl;
    hipMemcpy(result, freq_domain, complex_data_size_bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < results_size; i++) {
        std::cout << result[i].x << " + " << result[i].y << "j" << std::endl;
    }

    hipfftDestroy(plan);
    hipFree(signal);
    hipFree(freq_domain);
    delete[] result;

    return EXIT_SUCCESS;
}
