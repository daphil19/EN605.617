
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef struct
{
	unsigned int *firstInputCpu;
	unsigned int *secondInputCpu;
	unsigned int *firstInputGpu;
	unsigned int *secondInputGpu;
} INPUT_ARRAYS_T;

typedef struct
{
	unsigned int* cpuOutputBuffer;
	unsigned int *addResult;
	unsigned int *subtractResult;
	unsigned int *multResult;
	unsigned int *modResult;
} OUTPUT_ARRAYS_T;

typedef struct
{
	unsigned int totalThreads;
	unsigned int blockSize;
	unsigned int numBlocks;
	size_t dataSizeBytes;
} INPUT_PARAMS_T;

static const int RANDOM_RANGE = 4;
static const int CONST_ARRAY_SIZE = 8192;
static const int CONST_SIZE_BYTES = CONST_ARRAY_SIZE * sizeof(unsigned int);

__constant__ static unsigned int const_first_input[CONST_ARRAY_SIZE];
__constant__ static unsigned int const_second_input[CONST_ARRAY_SIZE];


__device__ unsigned int get_thread_index()
{
	return (blockIdx.x * blockDim.x) + threadIdx.x;
}


// in order to effectively leverage shared memory, we'll perform all operations at once
__global__ void perform_operations_shared(unsigned int* firstInput, unsigned int* secondInput, OUTPUT_ARRAYS_T output, unsigned int blockSize) {
    unsigned int idx = get_thread_index();

    // dynaic shared memory
    // NOTE that we have to put both sets of inputs into a single array... yay for index math!
    extern __shared__ unsigned int sharedInput[];

    // load inputs into shared memory
    sharedInput[idx] = firstInput[idx];

    // TODO do we need a protection here?
    sharedInput[idx + blockSize] = secondInput[idx];

	__syncthreads();

    // perform operations using shared memory
	output.addResult[idx] = sharedInput[idx] + sharedInput[idx + blockSize];
    output.subtractResult[idx] = sharedInput[idx] - sharedInput[idx + blockSize];
	output.multResult[idx] = sharedInput[idx] * sharedInput[idx + blockSize];
	output.modResult[idx] = sharedInput[idx] % sharedInput[idx + blockSize];

      __syncthreads();

}

__global__ void perform_operations_constant(OUTPUT_ARRAYS_T output) {
    unsigned int idx = get_thread_index();
	output.addResult[idx] = const_first_input[idx] + const_second_input[idx];
    output.subtractResult[idx] = const_first_input[idx] - const_second_input[idx];
    output.multResult[idx] = const_first_input[idx] * const_second_input[idx];
    output.modResult[idx] = const_first_input[idx] % const_second_input[idx];

}


__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

void initialize_inputs(INPUT_PARAMS_T *inputParams, INPUT_ARRAYS_T *input, OUTPUT_ARRAYS_T *output) {
	input->firstInputCpu = new unsigned int[inputParams->totalThreads];
	input->secondInputCpu = new unsigned int[inputParams->totalThreads];
	hipMalloc((void **)&(input->firstInputGpu), inputParams->dataSizeBytes);
	hipMalloc((void **)&(input->secondInputGpu), inputParams->dataSizeBytes);

	for (int i = 0; i < inputParams->totalThreads; i++) {
		input->firstInputCpu[i] = i;
		input->secondInputCpu[i] = rand() % RANDOM_RANGE;
	}

	hipMemcpy(input->firstInputGpu, input->firstInputCpu, inputParams->dataSizeBytes, hipMemcpyHostToDevice);
	hipMemcpy(input->secondInputGpu, input->secondInputCpu, inputParams->dataSizeBytes, hipMemcpyHostToDevice);

	output->cpuOutputBuffer = new unsigned int[inputParams->totalThreads];
	hipMalloc((void **)&(output->addResult), inputParams->dataSizeBytes);
	hipMalloc((void **)&(output->subtractResult), inputParams->dataSizeBytes);
	hipMalloc((void **)&(output->multResult), inputParams->dataSizeBytes);
	hipMalloc((void **)&(output->modResult), inputParams->dataSizeBytes);

}

void initialize_constant_inputs(INPUT_ARRAYS_T *input, OUTPUT_ARRAYS_T *output) {
	input->firstInputCpu = new unsigned int[CONST_ARRAY_SIZE];
	input->secondInputCpu = new unsigned int[CONST_ARRAY_SIZE];

	for (int i = 0; i < CONST_ARRAY_SIZE; i++) {
		input->firstInputCpu[i] = i;
		input->secondInputCpu[i] = rand() % RANDOM_RANGE;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(const_first_input), input->firstInputCpu, CONST_SIZE_BYTES);
	hipMemcpyToSymbol(HIP_SYMBOL(const_second_input), input->secondInputCpu, CONST_SIZE_BYTES);

	output->cpuOutputBuffer = new unsigned int[CONST_ARRAY_SIZE];
	hipMalloc((void **)&(output->addResult), CONST_SIZE_BYTES);
	hipMalloc((void **)&(output->subtractResult), CONST_SIZE_BYTES);
	hipMalloc((void **)&(output->multResult), CONST_SIZE_BYTES);
	hipMalloc((void **)&(output->modResult), CONST_SIZE_BYTES);


}

void cleanup(INPUT_ARRAYS_T *inputs, OUTPUT_ARRAYS_T* outputs) {
	delete[] inputs->firstInputCpu;
	delete[] inputs->secondInputCpu;
	hipFree(inputs->firstInputGpu);
	hipFree(inputs->secondInputGpu);

	delete[] outputs->cpuOutputBuffer;
	hipFree(outputs->addResult);
	hipFree(outputs->subtractResult);
	hipFree(outputs->multResult);
	hipFree(outputs->modResult);
}

void cleanup_const(INPUT_ARRAYS_T *inputs, OUTPUT_ARRAYS_T* outputs) {
	delete[] inputs->firstInputCpu;
	delete[] inputs->secondInputCpu;

	delete[] outputs->cpuOutputBuffer;
	hipFree(outputs->addResult);
	hipFree(outputs->subtractResult);
	hipFree(outputs->multResult);
	hipFree(outputs->modResult);

}


int main(int argc, char* *argv) {

	// read command line arguments
	unsigned int totalThreads = (1 << 20);
	unsigned int blockSize = 256;

	if (argc >= 2)
	{
		totalThreads = atoi(argv[1]);
	}
	else
	{
		printf("Using default total threads %d\n", totalThreads);
	}
	if (argc >= 3)
	{
		blockSize = atoi(argv[2]);
	}
	else
	{
		printf("Using default block size %d\n", blockSize);
	}

	unsigned int numBlocks = totalThreads / blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0)
	{
		++numBlocks;
		totalThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	size_t dataSizeBytes = sizeof(unsigned int) * totalThreads;

	INPUT_PARAMS_T inputParams = {
		totalThreads,
		blockSize,
		numBlocks,
		dataSizeBytes,
	};


	INPUT_ARRAYS_T input;
	OUTPUT_ARRAYS_T output;

    initialize_inputs(&inputParams, &input, &output);

	hipEvent_t start_time = get_time();
	// TODO I thought the shared memory size should be based on block size, but that seemed to break?
	perform_operations_shared<<<numBlocks, blockSize, totalThreads * sizeof(unsigned int) * 2>>>(input.firstInputGpu, input.secondInputGpu, output, blockSize);
	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);

	float sharedDelta = 0;
	hipEventElapsedTime(&sharedDelta, start_time, end_time);

	hipMemcpy(output.cpuOutputBuffer, output.addResult, dataSizeBytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < totalThreads; i++) {
		printf("%d + %d = %d\n", input.firstInputCpu[i], input.secondInputCpu[i], output.cpuOutputBuffer[i]);
	}

	hipMemcpy(output.cpuOutputBuffer, output.subtractResult, dataSizeBytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < totalThreads; i++) {
		printf("%d - %d = %d\n", input.firstInputCpu[i], input.secondInputCpu[i], output.cpuOutputBuffer[i]);
	}

	hipMemcpy(output.cpuOutputBuffer, output.multResult, dataSizeBytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < totalThreads; i++) {
		printf("%d * %d = %d\n", input.firstInputCpu[i], input.secondInputCpu[i], output.cpuOutputBuffer[i]);
	}

	hipMemcpy(output.cpuOutputBuffer, output.modResult, dataSizeBytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < totalThreads; i++) {
		printf("%d %% %d = %d\n", input.firstInputCpu[i], input.secondInputCpu[i], output.cpuOutputBuffer[i]);
	}

	cleanup(&input, &output);

	INPUT_ARRAYS_T constInput;
	OUTPUT_ARRAYS_T constOutput;

	initialize_constant_inputs(&constInput, &constOutput);

    start_time = get_time();
    perform_operations_constant<<<inputParams.numBlocks, CONST_ARRAY_SIZE / inputParams.numBlocks>>>(constOutput);
    end_time = get_time();
	hipEventSynchronize(end_time);

	float constantDelta = 0;
	hipEventElapsedTime(&constantDelta, start_time, end_time);


	hipMemcpy(constOutput.cpuOutputBuffer, constOutput.addResult, CONST_SIZE_BYTES, hipMemcpyDeviceToHost);
	for (int i = 0; i < CONST_ARRAY_SIZE; i++) {
		printf("%d + %d = %d\n", constInput.firstInputCpu[i], constInput.secondInputCpu[i], constOutput.cpuOutputBuffer[i]);
	}

	hipMemcpy(constOutput.cpuOutputBuffer, constOutput.subtractResult, CONST_SIZE_BYTES, hipMemcpyDeviceToHost);
	for (int i = 0; i < CONST_ARRAY_SIZE; i++) {
		printf("%d - %d = %d\n", constInput.firstInputCpu[i], constInput.secondInputCpu[i], constOutput.cpuOutputBuffer[i]);
	}

	hipMemcpy(constOutput.cpuOutputBuffer, constOutput.multResult, CONST_SIZE_BYTES, hipMemcpyDeviceToHost);
	for (int i = 0; i < CONST_ARRAY_SIZE; i++) {
		printf("%d * %d = %d\n", constInput.firstInputCpu[i], constInput.secondInputCpu[i], constOutput.cpuOutputBuffer[i]);
	}

	hipMemcpy(constOutput.cpuOutputBuffer, constOutput.modResult, CONST_SIZE_BYTES, hipMemcpyDeviceToHost);
	for (int i = 0; i < CONST_ARRAY_SIZE; i++) {
		printf("%d %% %d = %d\n", constInput.firstInputCpu[i], constInput.secondInputCpu[i], constOutput.cpuOutputBuffer[i]);
	}

	cleanup_const(&constInput, &constOutput);


    printf("Shared memory execution time: %f ms\n", sharedDelta);
    printf("Constant memory execution time: %f ms\n", constantDelta);

	return EXIT_SUCCESS;
}
