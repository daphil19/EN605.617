// Based on the work of Andrew Krepps

// NOTE formatting is based on the default formatter configuration of VS code

#include <stdio.h>
#include <stdlib.h>

#include "operations.cuh"

#define RANDOM_RANGE 4

// TODO if I had thought about this more I probably would have used a class instead of structs

typedef struct {
	unsigned int * firstInputCpu;
	unsigned int * secondInputCpu;
	unsigned int * firstInputGpu;
	unsigned int * secondInputGpu;
} INPUT_ARRAYS_T;

typedef struct {
	unsigned int *operationResultGpu;
	unsigned int *addResultCpu;
	unsigned int *subtractResultCpu;
	unsigned int *multResultCpu;
	unsigned int *modResultCpu;
} OUTPUT_ARRAYS_T;

typedef struct {
	unsigned int totalThreads;
	unsigned int blockSize;
	unsigned int numBlocks;
	size_t dataSizeBytes;
} INPUT_PARAMS_T;

void allocate_paged_inputs(INPUT_PARAMS_T* inputParams, INPUT_ARRAYS_T* input, OUTPUT_ARRAYS_T * output) {
		input->firstInputCpu = new unsigned int[inputParams->totalThreads];
		input->secondInputCpu = new unsigned int[inputParams->totalThreads]; 
		hipMalloc((void **)&(input->firstInputGpu), inputParams->dataSizeBytes);
		hipMalloc((void **)&(input->secondInputGpu), inputParams->dataSizeBytes);

		hipMalloc((void**)&(output->operationResultGpu), inputParams->dataSizeBytes);
		output->addResultCpu = new unsigned int[inputParams->totalThreads];
		output->subtractResultCpu = new unsigned int[inputParams->totalThreads];
		output->multResultCpu = new unsigned int[inputParams->totalThreads];
		output->modResultCpu = new unsigned int[inputParams->totalThreads];

}

void cleanup_paged_inputs(INPUT_ARRAYS_T* input, OUTPUT_ARRAYS_T* output) {
	delete[] input->firstInputCpu;
	delete[] input->secondInputCpu;
	hipFree(input->firstInputGpu);
	hipFree(input->secondInputGpu);

	hipFree(output->operationResultGpu);
	delete[] output->addResultCpu;
	delete[] output->subtractResultCpu;
	delete[] output->multResultCpu;
	delete[] output->modResultCpu;
}

void allocate_pinned_inputs(INPUT_PARAMS_T* inputParams, INPUT_ARRAYS_T* input, OUTPUT_ARRAYS_T* output) {
		hipHostMalloc((void **)&(input->firstInputCpu), inputParams->dataSizeBytes);
		hipHostMalloc((void **)&(input->secondInputCpu), inputParams->dataSizeBytes);
		hipMalloc((void **)&(input->firstInputGpu), inputParams->dataSizeBytes);
		hipMalloc((void **)&(input->secondInputGpu), inputParams->dataSizeBytes);

		hipMalloc((void**)&(output->operationResultGpu), inputParams->dataSizeBytes);
		hipHostMalloc((void **)&(output->addResultCpu), inputParams->dataSizeBytes);
		hipHostMalloc((void **)&(output->subtractResultCpu), inputParams->dataSizeBytes);
		hipHostMalloc((void **)&(output->multResultCpu), inputParams->dataSizeBytes);
		hipHostMalloc((void **)&(output->modResultCpu), inputParams->dataSizeBytes);
}

void cleanup_pinned_inputs(INPUT_ARRAYS_T* input, OUTPUT_ARRAYS_T* output) {
	hipHostFree(input->firstInputCpu);
	hipHostFree(input->secondInputCpu);
	hipFree(input->firstInputGpu);
	hipFree(input->secondInputGpu);

	hipFree(output->operationResultGpu);
	hipHostFree(output->addResultCpu);
	hipHostFree(output->subtractResultCpu);
	hipHostFree(output->multResultCpu);
	hipHostFree(output->modResultCpu);
}

void initialize_inputs(INPUT_PARAMS_T* inputParams, INPUT_ARRAYS_T* input) {
		for (int i = 0; i < inputParams->totalThreads; i++) {
		input->firstInputCpu[i] = i;
		input->secondInputCpu[i] = rand() % RANDOM_RANGE;
	}
}

__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

float perform_operations(INPUT_PARAMS_T * inputParams, INPUT_ARRAYS_T* input, OUTPUT_ARRAYS_T* output) {
	hipEvent_t start_time = get_time();


	hipMemcpy(input->firstInputGpu, input->firstInputCpu, inputParams->dataSizeBytes, hipMemcpyHostToDevice);
	hipMemcpy(input->secondInputGpu, input->secondInputCpu, inputParams->dataSizeBytes, hipMemcpyHostToDevice);


	add<<<inputParams->numBlocks, inputParams->blockSize>>>(output->operationResultGpu, input->firstInputGpu, input->secondInputGpu);

	hipMemcpy(output->addResultCpu, output->operationResultGpu, inputParams->dataSizeBytes, hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < inputParams->totalThreads; i++)
	{
		printf("%d + %d = %d\n", input->firstInputCpu[i], input->secondInputCpu[i], output->addResultCpu[i]);
	}

	subtract<<<inputParams->numBlocks, inputParams->blockSize>>>(output->operationResultGpu, input->firstInputGpu, input->secondInputGpu);
	hipMemcpy(output->subtractResultCpu, output->operationResultGpu, inputParams->dataSizeBytes, hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < inputParams->totalThreads; i++)
	{
		printf("%d - %d = %d\n", input->firstInputCpu[i], input->secondInputCpu[i], output->subtractResultCpu[i]);
	}

	mult<<<inputParams->numBlocks, inputParams->blockSize>>>(output->operationResultGpu, input->firstInputGpu, input->secondInputGpu);
	hipMemcpy(output->multResultCpu, output->operationResultGpu, inputParams->dataSizeBytes, hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < inputParams->totalThreads; i++)
	{
		printf("%d * %d = %d\n", input->firstInputCpu[i], input->secondInputCpu[i], output->multResultCpu[i]);
	}

	mod<<<inputParams->numBlocks, inputParams->blockSize>>>(output->operationResultGpu, input->firstInputGpu, input->secondInputGpu);
	hipMemcpy(output->modResultCpu, output->operationResultGpu, inputParams->dataSizeBytes, hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < inputParams->totalThreads; i++)
	{
		printf("%d %% %d = %d\n", input->firstInputCpu[i], input->secondInputCpu[i], output->modResultCpu[i]);
	}

		hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);

	float delta = 0;
	hipEventElapsedTime(&delta, start_time, end_time);
	return delta;

}

int main(int argc, char **argv)
{

	// read command line arguments
	unsigned int totalThreads = (1 << 20);
	unsigned int blockSize = 256;

	if (argc >= 2)
	{
		totalThreads = atoi(argv[1]);
	}
	else
	{
		printf("Using default total threads %d\n", totalThreads);
	}
	if (argc >= 3)
	{
		blockSize = atoi(argv[2]);
	}
	else
	{
		printf("Using default block size %d\n", blockSize);
	}

	unsigned int numBlocks = totalThreads / blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0)
	{
		++numBlocks;
		totalThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	size_t dataSizeBytes = sizeof(unsigned int) * totalThreads;

	INPUT_PARAMS_T inputParams = {
	totalThreads,
	blockSize,
	numBlocks,
	dataSizeBytes,
	};

	INPUT_ARRAYS_T input;

	OUTPUT_ARRAYS_T output;

	allocate_paged_inputs(&inputParams, &input, &output);
	initialize_inputs(&inputParams, &input);
	printf("Executing paged operations...");
	float pagedDelta = perform_operations(&inputParams, &input, &output);
	cleanup_paged_inputs(&input, &output);

	allocate_pinned_inputs(&inputParams, &input, &output);
	initialize_inputs(&inputParams, &input);
	printf("Executing pinned oeprations...");
	float pinnedDelta = perform_operations(&inputParams, &input, &output);
	cleanup_pinned_inputs(&input, &output);

	printf("Paged operations too %f ms\n", pagedDelta);
	printf("Pinned operations took %f ms\n", pinnedDelta);

	return EXIT_SUCCESS;
}
