
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
 * The maximum and minimum integer values of the range of printable characters
 * in the ASCII alphabet. Used by encrypt kernel to wrap adjust values to that
 * ciphertext is always printable.
 */
#define MAX_PRINTABLE 128
#define MIN_PRINTABLE 64

#define NUM_ALPHA MAX_PRINTABLE - MIN_PRINTABLE

__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

__global__ void encrypt(unsigned int *text, unsigned int *key, unsigned int *result)
{ /* Calculate the current index */
	const unsigned int
		idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	/*
	 * Adjust value of text and key to be based at 0
	 * Printable ASCII starts at MIN_PRINTABLE, but 0 start is easier to work with
	 */
	char adjusted_text = text[idx] - MIN_PRINTABLE;
	char adjusted_key = key[idx] - MIN_PRINTABLE;

	/* The cipher character is the text char added to the key char modulo the number of chars in the alphabet*/
	char cipherchar = (adjusted_text + adjusted_key - 1) % (NUM_ALPHA);

	/* adjust back to normal ascii (starting at MIN_PRINTABLE) and save to result */
	result[idx] = (unsigned int)cipherchar + MIN_PRINTABLE;
}

__global__ void decrypt(unsigned int *text, unsigned int *key, unsigned int *result)
{ /* Calculate the current index */
	const unsigned int
		idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	/*
	 * Adjust value of text and key to be based at 0
	 * Printable ASCII starts at MIN_PRINTABLE, but 0 start is easier to work with
	 */
	char adjusted_text = text[idx] - MIN_PRINTABLE;
	char adjusted_key = key[idx] - MIN_PRINTABLE;

	/* The cipher character is the text char added to the key char modulo the number of chars in the alphabet*/
	char cipherchar = (adjusted_text - (adjusted_key - 1)) % (NUM_ALPHA);

	/* adjust back to normal ascii (starting at MIN_PRINTABLE) and save to result */
	result[idx] = (unsigned int)cipherchar + MIN_PRINTABLE;
}

void print_encryption_results(unsigned int *text, unsigned int *key, unsigned int *result, unsigned int *decrypt, int size)
{
	for (int i = 0; i < size; i++)
	{
		printf("%c + %c -> %c -> %c\n", text[i], key[i], result[i], decrypt[i]);
	}
}

void pageable_transfer_execution(int array_size, int threads_per_block, FILE *input_fp, FILE *key_fp)
{ /* Calculate the size of the array
   */
	int array_size_in_bytes = (sizeof(unsigned int) * (array_size));

	unsigned int *cpu_text = (unsigned int *)malloc(array_size_in_bytes);
	unsigned int *cpu_key = (unsigned int *)malloc(array_size_in_bytes);
	unsigned int *cpu_result = (unsigned int *)malloc(array_size_in_bytes);
	unsigned int *cpu_decrypt = (unsigned int *)malloc(array_size_in_bytes);

	/* Read characters from the input and key files into the text and key arrays respectively */
	// Code left out for brevity sake
	char buf[array_size];
	// fgets size is the number of bytes to read PLUS the null terminator
	fgets(buf, array_size + 1, input_fp);
	// probably not the fastest way to do this, but it's the way that works in my mind
	for (int i = 0; i < array_size; i++)
	{
		cpu_text[i] = buf[i];
	}

	fgets(buf, array_size + 1, key_fp);
	for (int i = 0; i < array_size; i++)
	{
		cpu_key[i] = buf[i];
	}

	unsigned int *gpu_text;
	unsigned int *gpu_key;
	unsigned int *gpu_result;
	unsigned int *gpu_decrypt;

	hipMalloc((void **)&gpu_text, array_size_in_bytes);
	hipMalloc((void **)&gpu_key, array_size_in_bytes);
	hipMalloc((void **)&gpu_result, array_size_in_bytes);
	hipMalloc((void **)&gpu_decrypt, array_size_in_bytes);

	/* Copy the CPU memory to the GPU memory */
	hipMemcpy(gpu_text, cpu_text, array_size_in_bytes, hipMemcpyHostToDevice);
	hipMemcpy(gpu_key, cpu_key, array_size_in_bytes, hipMemcpyHostToDevice);

	/* Designate the number of blocks and threads */
	const unsigned int num_blocks = array_size / threads_per_block;
	const unsigned int num_threads = array_size / num_blocks;

	/* Execute the encryption kernel and keep track of start and end time for duration */
	float duration = 0;
	hipEvent_t start_time = get_time();

	encrypt<<<num_blocks, num_threads>>>(gpu_text, gpu_key, gpu_result);

	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

	/* Copy the changed GPU memory back to the CPU */
	hipMemcpy(cpu_result, gpu_result, array_size_in_bytes, hipMemcpyDeviceToHost);

	decrypt<<<num_blocks, num_threads>>>(gpu_result, gpu_key, gpu_decrypt);
	hipMemcpy(cpu_decrypt, gpu_decrypt, array_size_in_bytes, hipMemcpyDeviceToHost);

	printf("Pageable Transfer- Duration: %fmsn\n", duration);
	print_encryption_results(cpu_text, cpu_key, cpu_result, cpu_decrypt, array_size);

	/* Free the GPU memory */
	// INSERT CODE HERE
	hipFree(gpu_text);
	hipFree(gpu_key);
	hipFree(gpu_result);
	hipFree(gpu_decrypt);

	/* Free the CPU memory */
	// INSERT CODE HERE
	free(cpu_text);
	free(cpu_key);
	free(cpu_result);
	free(cpu_decrypt);
}

void pinned_transfer_execution(int array_size, int threads_per_block, FILE *input_fp, FILE *key_fp)
{ // Code left out for brevity sake
	int array_size_in_bytes = (sizeof(unsigned int) * (array_size));
	unsigned int *cpu_text_pinned;
	unsigned int *cpu_key_pinned;
	unsigned int *cpu_result_pinned;
	unsigned int *cpu_decrypt_pinned;

	// pin it
	hipHostMalloc((void **)&cpu_text_pinned, array_size_in_bytes, hipHostMallocDefault);
	hipHostMalloc((void **)&cpu_key_pinned, array_size_in_bytes, hipHostMallocDefault);
	hipHostMalloc((void **)&cpu_result_pinned, array_size_in_bytes, hipHostMallocDefault);
	hipHostMalloc((void **)&cpu_decrypt_pinned, array_size_in_bytes, hipHostMallocDefault);

	/* Copy the memory over */
	// INSERT CODE HERE
	char buf[array_size];
	fgets(buf, array_size + 1, input_fp);
	// probably not the fastest way to do this, but it's the way that works in my mind
	for (int i = 0; i < array_size; i++)
	{
		cpu_text_pinned[i] = buf[i];
	}

	fgets(buf, array_size + 1, key_fp);
	for (int i = 0; i < array_size; i++)
	{
		cpu_key_pinned[i] = buf[i];
	}

	/* Declare and allocate pointers for GPU based parameters */
	unsigned int *gpu_text;
	unsigned int *gpu_key;
	unsigned int *gpu_result;
	unsigned int *gpu_decrypt;

	hipMalloc((void **)&gpu_text, array_size_in_bytes);
	hipMalloc((void **)&gpu_key, array_size_in_bytes);
	hipMalloc((void **)&gpu_result, array_size_in_bytes);
	hipMalloc((void **)&gpu_decrypt, array_size_in_bytes);

	/* Copy the CPU memory to the GPU memory */
	hipMemcpy(gpu_text, cpu_text_pinned, array_size_in_bytes, hipMemcpyHostToDevice);
	hipMemcpy(gpu_key, cpu_key_pinned, array_size_in_bytes, hipMemcpyHostToDevice);

	/* Designate the number of blocks and threads */
	const unsigned int num_blocks = array_size / threads_per_block;
	const unsigned int num_threads = array_size / num_blocks;

	/* Execute the encryption kernel and keep track of start and end time for duration */
	float duration = 0;
	hipEvent_t start_time = get_time();

	encrypt<<<num_blocks, num_threads>>>(gpu_text, gpu_key, gpu_result);

	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	hipEventElapsedTime(&duration, start_time, end_time);

	/* Copy the changed GPU memory back to the CPU */
	hipMemcpy(cpu_result_pinned, gpu_result, array_size_in_bytes, hipMemcpyDeviceToHost);

	decrypt<<<num_blocks, num_threads>>>(gpu_result, gpu_key, gpu_decrypt);
	hipMemcpy(cpu_decrypt_pinned, gpu_decrypt, array_size_in_bytes, hipMemcpyDeviceToHost);

	printf("Pinned Transfer- Duration: %fmsn\n", duration);
	print_encryption_results(cpu_text_pinned, cpu_key_pinned, cpu_result_pinned, cpu_decrypt_pinned, array_size);

	/* Free the GPU memory */
	hipFree(gpu_text);
	hipFree(gpu_key);
	hipFree(gpu_result);
	hipFree(gpu_decrypt);

	/* Free the pinned CPU memory */
	hipHostFree(cpu_text_pinned);
	hipHostFree(cpu_key_pinned);
	hipHostFree(cpu_result_pinned);
	hipHostFree(cpu_decrypt_pinned);
}

/** * Prints the correct usage of this file * @name is the name of the executable (argv[0]) */
void print_usage(char *name)
{
	printf("Usage: %s <total_num_threads> <threads_per_block> <input_string> <input_offset>\n", name);
}

/**
 * Performs simple setup functions before calling the pageable_transfer_execution()
 * function. * Makes sure the files are valid, handles opening and closing of file pointers.
 */
void pageable_transfer(int num_threads, int threads_per_block, char *input_file, char *key_file)
{

	// Code left out for brevity sake

	FILE *input_fp = fopen(input_file, "r");
	FILE *key_fp = fopen(key_file, "r");

	/* Perform the pageable transfer */
	pageable_transfer_execution(num_threads, threads_per_block, input_fp, key_fp);

	fclose(input_fp);
	fclose(key_fp);
}

/**
 * Performs setup functions before calling the pageable_transfer_execution()
 * function.
 * Makes sure the files are valid, handles opening and closing of file pointers.
 */
void pinned_transfer(int num_threads, int threads_per_block, char *input_file, char *key_file)
{

	// Code left out for brevity sake

	FILE *input_fp = fopen(input_file, "r");
	FILE *key_fp = fopen(key_file, "r");

	/* Perform the pageable transfer */
	pinned_transfer_execution(num_threads, threads_per_block, input_fp, key_fp);

	fclose(input_fp);
	fclose(key_fp);
}

/**
 * Entry point for excution. Checks command line arguments and
 * opens input files, then passes execution to subordinate main_sub()
 */
int main(int argc, char *argv[])
{
	/* Check the number of arguments, print usage if wrong
	 */
	if (argc != 5)
	{
		printf("Error: Incorrect number of command line arguments\n");
		print_usage(argv[0]);
		exit(-1);
	}

	/* Check the values for num_threads and threads_per_block */
	int num_threads = atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);
	if (num_threads <= 0 || threads_per_block <= 0)
	{
		printf("Error: num_threads and threads_per_block must be integer > 0");
		print_usage(argv[0]);
		exit(-1);
	}

	if (threads_per_block > num_threads)
	{
		printf("Error: threads per block is greater than number of threads\n");
		print_usage(argv[0]);
		exit(-1);
	}

	printf("\n");
	/* Perform the pageable transfer */
	pageable_transfer(num_threads, threads_per_block, argv[3], argv[4]);

	printf("-----------------------------------------------------------------\n");

	/* Perform the pinned transfer */
	pinned_transfer(num_threads, threads_per_block, argv[3], argv[4]);

	return EXIT_SUCCESS;
}
