#include <stdio.h>
#include "operations.cuh"

static const int RANDOM_RANGE = 4;

// struct for handling input and output arrays for a given operation
typedef struct
{
    unsigned int *firstInput;
    unsigned int *secondInput;
    unsigned int *output;
} OPERATION_ARRAYS_T;

__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
    return time;
}

// print the delta based on the provided start and stop events
__host__ void print_delta(hipEvent_t start, hipEvent_t stop)
{
    hipEventSynchronize(stop);

    float delta = 0;
    hipEventElapsedTime(&delta, start, stop);
    printf("%f\n", delta);
}

// allocates **GPU** based arrays for an operation
__host__ OPERATION_ARRAYS_T initialize_operation_arrays(size_t dataSizeBytes)
{
    OPERATION_ARRAYS_T ops;
    hipMalloc((void **)&ops.firstInput, dataSizeBytes);
    hipMalloc((void **)&ops.secondInput, dataSizeBytes);
    hipMalloc((void **)&ops.output, dataSizeBytes);

    return ops;
}

// frees the **GPU** based arrays for an operation
__host__ void free_operation_arrays(OPERATION_ARRAYS_T ops)
{
    hipFree(ops.firstInput);
    hipFree(ops.secondInput);
    hipFree(ops.output);
}

// pretty-prints the results of a given set of operations inputs and output
__host__ void printResults(OPERATION_ARRAYS_T op, int size, char operation)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d %c %d = %d\n", op.firstInput[i], operation, op.secondInput[i], op.output[i]);
    }
}

int main(int argc, char **argv)
{
    // read command line arguments
    unsigned int totalThreads = (1 << 20);
    unsigned int blockSize = 256;

    if (argc >= 2)
    {
        totalThreads = atoi(argv[1]);
    }
    else
    {
        printf("Using default total threads %d\n", totalThreads);
    }
    if (argc >= 3)
    {
        blockSize = atoi(argv[2]);
    }
    else
    {
        printf("Using default block size %d\n", blockSize);
    }
    // "quiet" flag. If provided, only the timings will be printed to terminal
    bool quiet = argc >= 4 && strncmp(argv[3], "--quiet", 7) == 0;

    unsigned int numBlocks = totalThreads / blockSize;

    // validate command line arguments
    if (totalThreads % blockSize != 0)
    {
        ++numBlocks;
        totalThreads = numBlocks * blockSize;

        printf("Warning: Total thread count is not evenly divisible by the block size\n");
        printf("The total number of threads will be rounded up to %d\n", totalThreads);
    }

    size_t dataSizeBytes = sizeof(unsigned int) * totalThreads;

    // Initialize streams
    hipStream_t streamAdd, streamSub, streamMul, streamMod;
    hipStreamCreate(&streamAdd);
    hipStreamCreate(&streamSub);
    hipStreamCreate(&streamMul);
    hipStreamCreate(&streamMod);

    // define and allocate arryas
    unsigned int *firstInputCpu, *secondInputCpu, *outputCpu;
    unsigned int *firstInputGpu, *secondInputGpu, *outputGpu; // non-streamed processing

    hipHostAlloc(&firstInputCpu, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&secondInputCpu, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&outputCpu, dataSizeBytes, hipHostMallocDefault);

    OPERATION_ARRAYS_T addOps = initialize_operation_arrays(dataSizeBytes);
    OPERATION_ARRAYS_T subOps = initialize_operation_arrays(dataSizeBytes);
    OPERATION_ARRAYS_T mulOps = initialize_operation_arrays(dataSizeBytes);
    OPERATION_ARRAYS_T modOps = initialize_operation_arrays(dataSizeBytes);

    hipMalloc((void **)&firstInputGpu, dataSizeBytes);
    hipMalloc((void **)&secondInputGpu, dataSizeBytes);
    hipMalloc((void **)&outputGpu, dataSizeBytes);

    // initialize array inputs
    for (int i = 0; i < totalThreads; i++)
    {
        firstInputCpu[i] = i;
        secondInputCpu[i] = rand() % RANDOM_RANGE;
    }

    // hipEventCreate(&start)
    hipEvent_t streamStart = get_time();

    // copy data onto gpu
    hipMemcpyAsync(addOps.firstInput, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamAdd);
    hipMemcpyAsync(addOps.secondInput, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamAdd);

    hipMemcpyAsync(subOps.firstInput, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamSub);
    hipMemcpyAsync(subOps.secondInput, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamSub);

    hipMemcpyAsync(mulOps.firstInput, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMul);
    hipMemcpyAsync(mulOps.secondInput, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMul);

    hipMemcpyAsync(modOps.firstInput, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMod);
    hipMemcpyAsync(modOps.secondInput, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMod);

    // block before we process, just to make sure the copy is done in time
    hipStreamSynchronize(streamAdd);
    add<<<numBlocks, blockSize, 0, streamAdd>>>(addOps.output, addOps.firstInput, addOps.secondInput);
    hipStreamSynchronize(streamAdd);

    hipStreamSynchronize(streamSub);
    subtract<<<numBlocks, blockSize, 0, streamAdd>>>(subOps.output, subOps.firstInput, subOps.secondInput);
    hipStreamSynchronize(streamSub);

    hipStreamSynchronize(streamMul);
    mult<<<numBlocks, blockSize, 0, streamAdd>>>(mulOps.output, mulOps.firstInput, mulOps.secondInput);
    hipStreamSynchronize(streamMul);

    hipStreamSynchronize(streamMod);
    mod<<<numBlocks, blockSize, 0, streamAdd>>>(modOps.output, modOps.firstInput, modOps.secondInput);
    hipStreamSynchronize(streamMod);

    hipEvent_t streamStop = get_time();

    if (!quiet)
    {
        OPERATION_ARRAYS_T host_outputs = {
            firstInputCpu,
            secondInputCpu,
            outputCpu};
        hipMemcpyAsync(host_outputs.output, addOps.output, dataSizeBytes, hipMemcpyHostToDevice, streamAdd);
        // block to make sure the copy is done correctly
        hipStreamSynchronize(streamAdd);
        printResults(host_outputs, totalThreads, '+');

        hipMemcpyAsync(host_outputs.output, subOps.output, dataSizeBytes, hipMemcpyHostToDevice, streamSub);
        hipStreamSynchronize(streamSub);
        printResults(host_outputs, totalThreads, '-');

        hipMemcpyAsync(host_outputs.output, mulOps.output, dataSizeBytes, hipMemcpyHostToDevice, streamMul);
        hipStreamSynchronize(streamMul);
        printResults(host_outputs, totalThreads, '*');

        hipMemcpyAsync(host_outputs.output, modOps.output, dataSizeBytes, hipMemcpyHostToDevice, streamMod);
        hipStreamSynchronize(streamMod);
        printResults(host_outputs, totalThreads, '%');
    }

    printf("stream runtime: ");
    print_delta(streamStart, streamStop);

    hipEvent_t syncStart = get_time();

    // yes, these copies are redundant, but their purpose is to emulate the operations above to provide a benchmark for performance
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    add<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    subtract<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    mult<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    mod<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);

    // we won't output the synchronous operations because we already know those work correctly

    hipEvent_t syncStop = get_time();
    printf("synchronous runtime: ");
    print_delta(syncStart, syncStop);

    hipHostFree(firstInputCpu);
    hipHostFree(secondInputCpu);
    hipHostFree(outputCpu);

    free_operation_arrays(addOps);
    free_operation_arrays(subOps);
    free_operation_arrays(mulOps);
    free_operation_arrays(modOps);

    hipFree(firstInputGpu);
    hipFree(secondInputGpu);
    hipFree(outputGpu);

    return EXIT_SUCCESS;
}
