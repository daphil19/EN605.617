#include<stdio.h>
#include "operations.cuh"

typedef struct {
    unsigned int *firstInput;
    unsigned int *secondInput;
    unsigned int *output;
} OPERATION_ARRAYS_T;

__host__ hipEvent_t get_time(void)
{
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

__host__ void print_delta(hipEvent_t start, hipEvent_t stop) {
    // TODO
    hipEventSynchronize(stop);

	float delta = 0;
	hipEventElapsedTime(&delta, start, stop);
    printf("%f\n", delta);
}

int main(int argc, char **argv)
{
	// read command line arguments
	unsigned int totalThreads = (1 << 20);
	unsigned int blockSize = 256;

	if (argc >= 2)
	{
		totalThreads = atoi(argv[1]);
	}
	else
	{
		printf("Using default total threads %d\n", totalThreads);
	}
	if (argc >= 3)
	{
		blockSize = atoi(argv[2]);
	}
	else
	{
		printf("Using default block size %d\n", blockSize);
	}

	unsigned int numBlocks = totalThreads / blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0)
	{
		++numBlocks;
		totalThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	size_t dataSizeBytes = sizeof(unsigned int) * totalThreads;

    // Initialize streams
    hipStream_t streamAdd, streamSub, streamMul, streamMod;
    hipStreamCreate(&streamAdd);
    hipStreamCreate(&streamSub);
    hipStreamCreate(&streamMul);
    hipStreamCreate(&streamMod);

    // initialize timing events
    // hipEvent_t startStream, stopStream, startPage, stopPage;


    // define and allocate arryas
    // TODO putting all of this into an array may make the most sense...
    unsigned int *firstInputCpu, *secondInputCpu;
    unsigned int *firstInputGpuAdd, *secondInputGpuAdd, *outputGpuAdd;
    unsigned int *firstInputGpuSub, *secondInputGpuSub, *outputGpuSub;
    unsigned int *firstInputGpuMul, *secondInputGpuMul, *outputGpuMul;
    unsigned int *firstInputGpuMod, *secondInputGpuMod, *outputGpuMod;
    unsigned int *firstInputGpu, *secondInputGpu, *outputGpu; // non-streamed processing

    hipHostAlloc(&firstInputCpu, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&secondInputCpu, dataSizeBytes, hipHostMallocDefault);

    hipMalloc((void**)&firstInputGpuAdd, dataSizeBytes);
    hipMalloc((void**)&secondInputGpuAdd, dataSizeBytes);
    hipMalloc((void**)&outputGpuAdd, dataSizeBytes);

    hipMalloc((void**)&firstInputGpuSub, dataSizeBytes);
    hipMalloc((void**)&secondInputGpuSub, dataSizeBytes);
    hipMalloc((void**)&outputGpuSub, dataSizeBytes);

    hipMalloc((void**)&firstInputGpuMul, dataSizeBytes);
    hipMalloc((void**)&secondInputGpuMul, dataSizeBytes);
    hipMalloc((void**)&outputGpuMul, dataSizeBytes);

    hipMalloc((void**)&firstInputGpuMod, dataSizeBytes);
    hipMalloc((void**)&secondInputGpuMod, dataSizeBytes);
    hipMalloc((void**)&outputGpuMod, dataSizeBytes);

    hipMalloc((void**)&firstInputGpu, dataSizeBytes);
    hipMalloc((void**)&secondInputGpu, dataSizeBytes);
    hipMalloc((void**)&outputGpu, dataSizeBytes);


    // initialize array inputs
    for (int i = 0; i < totalThreads; i++) {
        firstInputCpu[i] = i;
        secondInputCpu[i] = rand();
    }

    // hipEventCreate(&start)
    hipEvent_t streamStart = get_time();

    // copy data onto gpu
    // TODO apparantly we async copy to the stream and then call the kernel
    hipMemcpyAsync(firstInputGpuAdd, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamAdd);
    hipMemcpyAsync(secondInputGpuAdd, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamAdd);

    hipMemcpyAsync(firstInputGpuSub, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamSub);
    hipMemcpyAsync(secondInputGpuSub, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamSub);

    hipMemcpyAsync(firstInputGpuMul, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMul);
    hipMemcpyAsync(secondInputGpuMul, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMul);

    hipMemcpyAsync(firstInputGpuMod, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMod);
    hipMemcpyAsync(secondInputGpuMod, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice, streamMod);


    // block before we process, just to make sure the copy is done in time
    hipStreamSynchronize(streamAdd);
    add<<<numBlocks, blockSize, 0, streamAdd>>>(outputGpuAdd, firstInputGpuAdd, secondInputGpuAdd);
    hipStreamSynchronize(streamAdd);

    hipStreamSynchronize(streamSub);
    subtract<<<numBlocks, blockSize, 0, streamAdd>>>(outputGpuAdd, firstInputGpuAdd, secondInputGpuAdd);
    hipStreamSynchronize(streamSub);

    hipStreamSynchronize(streamMul);
    mult<<<numBlocks, blockSize, 0, streamAdd>>>(outputGpuAdd, firstInputGpuAdd, secondInputGpuAdd);
    hipStreamSynchronize(streamMul);

    hipStreamSynchronize(streamMod);
    mod<<<numBlocks, blockSize, 0, streamAdd>>>(outputGpuAdd, firstInputGpuAdd, secondInputGpuAdd);
    hipStreamSynchronize(streamMod);

    hipEvent_t streamStop = get_time();
    print_delta(streamStart, streamStop);

    // TODO output!

    hipEvent_t syncStart = get_time();

    // yes, these copies are redundant, but their purpose is to emulate the operations above to provide a benchmark for performance
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    add<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    subtract<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    mult<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    mod<<<numBlocks, blockSize>>>(outputGpu, firstInputGpu, secondInputGpu);
    hipMemcpy(firstInputGpu, firstInputCpu, dataSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(secondInputGpu, secondInputCpu, dataSizeBytes, hipMemcpyHostToDevice);

    hipEvent_t syncStop = get_time();
    print_delta(syncStart, syncStop);


    hipHostFree(firstInputCpu);
    hipHostFree(secondInputCpu);

    hipFree(firstInputGpuAdd);
    hipFree(secondInputGpuAdd);
    hipFree(outputGpuAdd);

    hipFree(firstInputGpuSub);
    hipFree(secondInputGpuSub);
    hipFree(outputGpuSub);

    hipFree(firstInputGpuMul);
    hipFree(secondInputGpuMul);
    hipFree(outputGpuMul);

    hipFree(firstInputGpuMod);
    hipFree(secondInputGpuMod);
    hipFree(outputGpuMod);

    hipFree(firstInputGpu);
    hipFree(secondInputGpu);
    hipFree(outputGpu);

    return EXIT_SUCCESS;
}
